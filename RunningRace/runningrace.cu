#include "hip/hip_runtime.h"
﻿#include <vector>
#include <cstdlib>
#include <assert.h>
#include <stdlib.h>
#include <stddef.h>
#include <time.h>
#include <stdio.h>
#include <hiprand.h>
#include <iostream>
#include <random>
#include <thread>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



// Constants
const int NUM_RUNNERS = 100;
const int RUNWAY_LENGTH = 100;
__device__ int order = 0;
__device__ int oneRunnerHasReached = 0;

// Runner class
class Runner {
public:
    // Constructor
    Runner() : code_(0), position_(0.0f), speed_(0.0f), final_order_(order) { }

    int getCode() const { return code_; }
    void setCode(int code) { code_ = code; }

    // Getter and setter for position
    float getPosition() const { return position_; }
    void setPosition(float position) { position_ = position; }

    // Getter and setter for speed
    float getSpeed() const { return speed_; }
    void setSpeed(float speed) { speed_ = speed; }

    int getFinalOrder() const { return final_order_; }
    void setFinalOrder(int final_order) { final_order_ = final_order; }
    // Update the runner's position based on their speed
    __host__ __device__ void updatePosition() {
        // Calculate the new position
        float newPosition = position_ + speed_;
        // Check if the runner has reached the end of the runway
        if (position_ >= 100) {
            position_ = 100;
        }
        else {
            if (newPosition >= RUNWAY_LENGTH) {
                order++;
                // Set the position to the end of the runway
                position_ = RUNWAY_LENGTH;
                final_order_ = order;

                //std::cout << "Runner " << final_order_ << " has finished the race at position " << position_ << "m" << std::endl;
            }
            else {
                // Update the position
                position_ = newPosition;
            }
        }



    }

private:
    int code_;       // The code of the runner
    float position_;  // The current position of the runner
    float speed_;     // The current speed of the runner
    int final_order_; //// The final position of the runner
};

// Random number generator
std::mt19937 rng;

// Function to generate a random speed for a runner
float generateRandomSpeed() {
    // Create a uniform distribution in the range [1, 5]
    std::uniform_real_distribution<float> dist(1.0f, 5.0f);

    // Generate and return a random number
    return dist(rng);
}
//Function to sort runners by current positions
bool compareRunnersByCurrentPositions(const Runner& a, const Runner& b) {
    return a.getPosition() > b.getPosition();
}
//Function to sort runners by final positions
bool compareRunnersByFinalOrder(const Runner& a, const Runner& b) {
    return a.getFinalOrder() < b.getFinalOrder();
}

// Function to print the positions of all runners
void printRunnerPositions(const Runner* runners) {
    // Print the position of each runner
    int finishedRunner = 0;
    for (int i = 0; i < NUM_RUNNERS; i++) {
        // Check if the current runner has reached the end of the runway
        if (runners[i].getPosition() >= RUNWAY_LENGTH && runners[i].getFinalOrder() == 1 && oneRunnerHasReached == 0) {
            finishedRunner = 1;

            std::cout << "\nRunner " << runners[i].getCode() << " has reached the end of the runway for the first time!" << std::endl;

        }
    }
    if (finishedRunner == 1) {
        std::cout << std::endl;

        std::cout << "Runner Code: \t Runner Location:" << std::endl;

        for (int i = 0; i < NUM_RUNNERS; i++) {
            oneRunnerHasReached++;
            std::cout << "Runner " << runners[i].getCode() << ":\t " << runners[i].getPosition() << "m" << std::endl;

        }
        std::cout << std::endl;
    }



    std::cout << "The race is in progress! " << std::endl;


}


// Function to print the positions of all runners
void printRunnerPositionsAfterAllFinishedTheRace(const Runner* runners) {
    std::cout << "\nThe race is over.\nHere are the results:\n" << std::endl;
    std::cout << "Runner Code: \t Runner Position:\n" << std::endl;

    for (int i = 0; i < NUM_RUNNERS; i++) {
        std::cout << "Runner " << runners[i].getCode() << ": " << runners[i].getPosition() << "m\t\t" << runners[i].getFinalOrder() << std::endl;
    }

    std::cout << std::endl;
}

// CUDA kernel to update the positions of all runners
__global__ void updateRunnerPositions(Runner* runners) {
    // Get the index of the current thread
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if the current thread is within the range of runners
    if (threadIndex < NUM_RUNNERS) {
        // Update the position of the current runner
        runners[threadIndex].updatePosition();
    }
}

int main() {
    std::cout << "A 100m running race is held in which 100 runners participate.\nRunners have instant position and instant speed.\nThis speed changes randomly between a minimum of 1 meter/second and a maximum of 5 meters/second.\nEach runner is calculated in parallel by different Threads on the graphics card.\nThe graphics card runs once per second, and the new instantaneous position of all runners is calculated at each run.\nWhen the first runner reaches the finish line, the current position of all runners is printed sequentially.\nWhen all runners finish the race, the ranking of the race is printed on the screen.\n" << std::endl;

    // Seed the random number generator
    rng.seed(std::random_device()());

    // Allocate memory for the runners on the device
    Runner* deviceRunners;
    hipMalloc(&deviceRunners, sizeof(Runner) * NUM_RUNNERS);

    // Allocate memory for the runners on the host
    Runner* hostRunners = new Runner[NUM_RUNNERS];

    for (int i = 0; i < NUM_RUNNERS; i++) {
        hostRunners[i].setCode(i + 1);
        hostRunners[i].setPosition(0.0f);
        hostRunners[i].setSpeed(0.0f);
        hostRunners[i].setFinalOrder(0);
        //hostRunners[i].setSpeed(generateRandomSpeed());

    }

    // Loop until all runners have reached the end of the runway
    bool allRunnersFinished = false;
    while (!allRunnersFinished) {
        //For everytime generated random speed for all runners 
        // Initialize the runners
        for (int i = 0; i < NUM_RUNNERS; i++) {
            hostRunners[i].setSpeed(generateRandomSpeed());
        }

        // Copy the host runners to the device
        hipMemcpy(deviceRunners, hostRunners, sizeof(Runner) * NUM_RUNNERS, hipMemcpyHostToDevice);

        // Launch the CUDA kernel to update the positions of the runners
        updateRunnerPositions << <1, NUM_RUNNERS >> > (deviceRunners);

        // Copy the updated runners back to the host
        hipMemcpy(hostRunners, deviceRunners, sizeof(Runner) * NUM_RUNNERS, hipMemcpyDeviceToHost);

        // Check if all of the runners have finished the race
        allRunnersFinished = true;
        for (int i = 0; i < NUM_RUNNERS; i++) {
            if (hostRunners[i].getPosition() < RUNWAY_LENGTH) {
                allRunnersFinished = false;
                break;
            }
        }
        std::this_thread::sleep_for(std::chrono::seconds(1));

        //Sort all runners by current positions.
        std::sort(hostRunners, hostRunners + NUM_RUNNERS, compareRunnersByCurrentPositions);

        // Print the positions of the runners
        printRunnerPositions(hostRunners);
    }

    //Sort all runners by final positions.
    std::sort(hostRunners, hostRunners + NUM_RUNNERS, compareRunnersByFinalOrder);
    //Print all runners positions after the race.
    printRunnerPositionsAfterAllFinishedTheRace(hostRunners);

    // Free the memory allocated for the runners on the device and host
    hipFree(deviceRunners);
    delete[] hostRunners;

    return 0;
}
